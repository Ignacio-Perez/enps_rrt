#include <hip/hip_runtime.h>
#include <stdio.h>
#include <include/hipcub/hipcub.hpp>

#define N 5



int main()
{
	float * val;
	float * d_val;
	float * d_val2;
	float * d_val3;
	int pos;

	val = new float[N];
	hipMalloc(&d_val,N*sizeof(float));
	hipMalloc(&d_val2,N*sizeof(float));
	hipMalloc(&d_val3,N*sizeof(float));

	for (int i =0; i< N; i++)
		val[i]=0.1 * (i+12);	
	val[N-1] = 0.004;
	hipMemcpy(d_val,val,N*sizeof(float),hipMemcpyHostToDevice);
	for (int i =0; i< N; i++) printf("val[%i]=%f\n",i,val[i]);

	for (int i =0; i< N; i++)
		val[i]=0.1 * (i+2);
	hipMemcpy(d_val2,val,N*sizeof(float),hipMemcpyHostToDevice);
	for (int i =0; i< N; i++) printf("val[%i]=%f\n",i,val[i]);
	size_t bytes;
	hipcub::DeviceReduce::Min(d_val3, bytes, d_val, d_val2, N);
 
	float res;
	hipMemcpy(&res,d_val2,sizeof(float),hipMemcpyDeviceToHost);

	printf("\n\n Minval=%f\n",res);

	hipFree(d_val);
	hipFree(d_val2);
	hipFree(d_val3);
	free(val);



}
