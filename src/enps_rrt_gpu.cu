#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

extern "C" { // nvcc compiles en C++
#include <enps_rrt.h>
#include <pgm.h>
}

#include <omp.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

typedef unsigned int uint;

using namespace std;
using namespace hipcub;

struct _dev_pointers {
	float *da,*db;
	float *dx,*dy,*dpx,*dpy,*dd,*ddp;
	float *dm, *dx_nearest, *dy_nearest, *dx_new, *dy_new;
	// for RRT*
	float *dc, *dcp, *ddpp;
	// for CUB
	void  *dcubtemp;
	hipcub::KeyValuePair<int,float> *d_argmin;
	int *dsegments;
	size_t temp_bytes;
	// for kernel optimization
	int num_multiproc;
} devp;

#define MAX(a,b) (a>b?a:b)
#define MIN(a,b) (a<b?a:b)
#define THRES_GPU 256

void init_params(const char* file, int n, float delta, int debug, int algorithm, RRT_PARAMS* params)
{
	PGM* map = load_pgm(file);
	params->map = load_pgm(file);

	remove_inner_obstacles(map);
	
	params->epsilon = ROBOT_RADIUS * ROBOT_RADIUS;
	params->delta =   delta;
	
	params->n = n;
	params->N = 1<<n; // Number of nodes in RRT
	
	params->debug = debug;
	
	params->p = map->width * RESOLUTION;
	params->q = map->height * RESOLUTION;
	
	params->algorithm = algorithm;
	
	float x=0,y=0;
	int c=0;
	params->a = (float*)malloc(map->width * map->height * sizeof(float));
	params->b = (float*)malloc(map->width * map->height * sizeof(float));
	
	for (int i=0;i<map->height;i++) {
		for (int j=0;j<map->width;j++) {
			x+=RESOLUTION;
			if (IS_OBSTACLE(map,i,j)){
				params->a[c] = x;
				params->b[c] = y;
				c++;
			}
		}
		y += RESOLUTION;
		x = 0;
	}
	params->m = 0;
	params->M = 1;
	
	while (params->M < c) {
		params->m++;
		params->M <<= 1;
	}
	if (debug) {
		printf("Map: %s\n",file);
		printf("Number of obstacles: %d\n",c);
		printf("Number of nodes: %d\n",params->M);
	}
	params->a = (float*)realloc(params->a, (params->M)*sizeof(float));
	params->b = (float*)realloc(params->b, (params->M)*sizeof(float));
	//free(params->a);
	//free(params->b);
	//hipMallocManaged(&params->a,(params->M)*sizeof(float));
	//hipMallocManaged(&params->b,(params->M)*sizeof(float));
	for (int i=c;i<params->M;i++) {
		params->a[i] = 3* params->p;
		params->b[i] = 3* params->q;
	}
	
	params->device = -1;
  	hipGetDevice(&params->device);
	struct hipDeviceProp_t devprop;
	hipGetDeviceProperties(&devprop,params->device);
 	printf("Using GPU: %s\n",devprop.name);
	devp.num_multiproc = devprop.multiProcessorCount;

	hipMalloc(&devp.da,(params->M)*sizeof(float));
	hipMalloc(&devp.db,(params->M)*sizeof(float));
	hipMemcpy(devp.da,params->a,(params->M)*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(devp.db,params->b,(params->M)*sizeof(float),hipMemcpyHostToDevice);

	destroy_pgm(map);
}

// initializes a vector dv with values val, from ini to end. At position ini, the value valini is given.
__global__ void k_init_vec(float* dv, float val, float valini, uint ini, uint end) 
{
	for(uint i = threadIdx.x+blockDim.x*blockIdx.x; i<end; i+=gridDim.x*blockDim.x) {
		if (i == ini)
			dv[i] = valini;
		else if (i > ini)
			dv[i] = val;
	}
}

void init_vars(float x_init, float y_init, const RRT_PARAMS* params, RRT_VARS* vars)
{
	vars->x = (float*)malloc((params->N)*sizeof(float));
	vars->y = (float*)malloc((params->N)*sizeof(float));
	
	vars->x[0] = x_init;
	vars->y[0] = y_init;
	
	for (int i=1; i< params->N; i++) {
		vars->x[i] = 3* params->p;
		vars->y[i] = 3* params->q;
	}

	CubDebugExit(hipMalloc(&devp.dx,(params->N)*sizeof(float)));
	CubDebugExit(hipMalloc(&devp.dy,(params->N)*sizeof(float)));
	CubDebugExit(hipMalloc(&devp.dpx,(params->N)*sizeof(float)));
	CubDebugExit(hipMalloc(&devp.dpy,(params->N)*sizeof(float)));
	k_init_vec<<<devp.num_multiproc*64,256>>>(devp.dx,3*params->p,x_init,0,params->N);
	k_init_vec<<<devp.num_multiproc*64,256>>>(devp.dy,3*params->q,y_init,0,params->N);
	
	vars->px = (float*)malloc((params->N)*sizeof(float));
	vars->py = (float*)malloc((params->N)*sizeof(float));
	
	vars->d = (float*)malloc((params->N)*sizeof(float));
	CubDebugExit(hipMalloc(&devp.dd,(params->N)*sizeof(float)));
			
	vars->dp = (float*)malloc((params->M)*sizeof(float));
	CubDebugExit(hipMalloc(&devp.ddp,(params->M)*sizeof(float)));

	hipMalloc(&devp.d_argmin,sizeof(KeyValuePair<int, float>));
	CubDebugExit(hipcub::DeviceReduce::ArgMin(NULL, devp.temp_bytes, devp.ddp, devp.d_argmin, MAX(params->M,params->N)));

	//CubDebugExit(hipMalloc(&devp.dcubtemp,devp.temp_bytes));
	CubDebugExit(hipMalloc(&devp.dm,sizeof(float)));
	CubDebugExit(hipMalloc(&devp.dx_nearest,sizeof(float)));
	CubDebugExit(hipMalloc(&devp.dy_nearest,sizeof(float)));
	CubDebugExit(hipMalloc(&devp.dx_new,sizeof(float)));
	CubDebugExit(hipMalloc(&devp.dy_new,sizeof(float)));

	vars->x_rand = 0;
	vars->y_rand = 0;
	
	vars->x_new = 0;
	vars->y_new = 0;
	
	vars->x_nearest = 0;
	vars->y_nearest = 0;
	
	vars->collision = 0;
	
	vars->index = 1;
	vars->halt = 0;
	
	if (params->algorithm == RRT_STAR_ALGORITHM) {
		vars->dpp = (float*)malloc(params->M * params->N * sizeof(float));
		CubDebugExit(hipMalloc(&devp.ddpp,(params->N*params->M)*sizeof(float)));
		vars->c = (float*)malloc(params->N * sizeof(float));
		CubDebugExit(hipMalloc(&devp.dc,(params->N)*sizeof(float)));
		vars->cp = (float*)malloc(params->N * sizeof(float));
		CubDebugExit(hipMalloc(&devp.dcp,(params->N)*sizeof(float)));
		vars->c[0] = 0;
		CubDebugExit(hipMemset(devp.dc,0,sizeof(float)));

		CubDebugExit(hipMalloc(&devp.dsegments,(params->N+1)*sizeof(float)));
		uint* segments = new uint[params->N+1];
		segments[0]=0;	
		for (int i=1;i<=params->N;i++)
			segments[i]=segments[i-1]+params->M;
		hipMemcpy(devp.dsegments,segments,(params->N+1)*sizeof(float),hipMemcpyHostToDevice);
		delete []segments;	

		size_t bytesSegmented;
		CubDebugExit(hipcub::DeviceSegmentedReduce::Min(NULL, bytesSegmented, devp.ddp, devp.dd, params->N, devp.dd, devp.dd + 1));	
		devp.temp_bytes=MAX(devp.temp_bytes,bytesSegmented);
	} 

	CubDebugExit(hipMalloc(&devp.dcubtemp,devp.temp_bytes));
}

void free_memory(RRT_PARAMS* params,RRT_VARS* vars)
{
	destroy_pgm(params->map);
	free(params->a);
	free(params->b);
	free(vars->x);
	free(vars->y);
	free(vars->px);
	free(vars->py);
	free(vars->d);
	free(vars->dp);

	//TODO: release GPU memory, see pointers devp
	hipFree(devp.da);
	hipFree(devp.db);
	hipFree(devp.dx);
	hipFree(devp.dy);
	hipFree(devp.dpx);
	hipFree(devp.dpy);
	hipFree(devp.dd);
	hipFree(devp.ddp);
	hipFree(devp.dcubtemp);
	hipFree(devp.dm);
	hipFree(devp.dx_nearest);
	hipFree(devp.dy_nearest);
	hipFree(devp.dx_new);
	hipFree(devp.dy_new);
	
	if (params->algorithm==RRT_STAR_ALGORITHM) {
		free(vars->dpp);
		free(vars->c);
		free(vars->cp);
	}
}


float rnd()
{
	return (float)rand()/(float)(RAND_MAX);
}


XYD xyd_min2(XYD a, XYD b)
{
	return a.d < b.d ? a : b;
}

#pragma omp declare reduction(xyd_min : XYD : omp_out=xyd_min2(omp_out,omp_in))\
		initializer(omp_priv={0,0,INF})

__global__ void k_sqdist(float *dd, float *dx, float *dy, float x_rand, float y_rand, uint index)
{
    for(uint i = threadIdx.x+blockDim.x*blockIdx.x; i<index; i+=gridDim.x*blockDim.x)
		dd[i] = (dx[i] - x_rand) * (dx[i] - x_rand) + (dy[i] - y_rand) * (dy[i] - y_rand);
}

__global__ void k_nearest(float *dd, float *dx_nearest, float *dy_nearest, float *dx_new, float *dy_new, float *dx, float *dy, KeyValuePair<int,float>* d_argmin, float x_rand, float y_rand, float delta) 
{
	float x_nearest = dx[d_argmin->key];
	float y_nearest = dy[d_argmin->key];
	float dmin = d_argmin->value;
	dx_nearest[0] = x_nearest;
	dy_nearest[0] = y_nearest;
	dd[0] = dmin;
	dx_new[0] = x_nearest + delta * (x_rand - x_nearest) / sqrtf(dmin);
	dy_new[0] = y_nearest + delta * (y_rand - y_nearest) / sqrtf(dmin);
}

void nearest(RRT_PARAMS* params, RRT_VARS* vars)
{
	if (vars->index > THRES_GPU) {
		// compute squared distances from all points in RRT to (x_rand,y_rand)
		k_sqdist<<<MIN(vars->index/256+1,devp.num_multiproc*8),256>>>(devp.dd, devp.dx, devp.dy, vars->x_rand, vars->y_rand, vars->index);
		CubDebugExit(hipDeviceSynchronize());
		
		hipcub::KeyValuePair<int, float> argmin;
		// compute minimun distance and nearest point	
		CubDebugExit(hipcub::DeviceReduce::ArgMin(devp.dcubtemp, devp.temp_bytes, devp.dd, devp.d_argmin, vars->index));
		CubDebugExit(hipDeviceSynchronize());

		// avoid copying the info to the CPU, all is done on the GPU
		k_nearest<<<1,1>>>(devp.dd, devp.dx_nearest, devp.dy_nearest, devp.dx_new, devp.dy_new, devp.dx, devp.dy, devp.d_argmin, vars->x_rand, vars->y_rand, params->delta);
		CubDebugExit(hipDeviceSynchronize());
		/*CubDebugExit(hipMemcpy(&argmin,devp.d_argmin,sizeof(hipcub::KeyValuePair<int,float>),hipMemcpyDeviceToHost));
		vars->d[0] = argmin.value;
		vars->x_nearest = vars->x[argmin.key];
		vars->y_nearest = vars->y[argmin.key];*/
	}
	else {
		// CPU version:
		// compute squared distances from all points in RRT to (x_rand,y_rand)
		#pragma omp parallel for
		for (int i=0;i<vars->index;i++) {
			vars->d[i] = (vars->x[i] - vars->x_rand) * (vars->x[i] - vars->x_rand) +
							(vars->y[i] - vars->y_rand) * (vars->y[i] - vars->y_rand);
		}

		// compute minimun distance and nearest point
		XYD value = {0,0,INF};
		#pragma omp parallel for reduction(xyd_min:value)
		for (int i=0;i<vars->index;i++) {
			XYD new_value = {vars->x[i],vars->y[i],vars->d[i]};
			value = xyd_min2(value,new_value);
		}
		vars->x_nearest = value.x;
		vars->y_nearest = value.y;
		vars->d[0] = value.d;
	}
}


// Squared distance from point (Cx,Cy) to segment [(Ax,Ay),(Bx,By)]
float p_dist(float Cx, float Cy, float Ax, float Ay, float Bx, float By)
{
	float u = (Cx-Ax)*(Bx-Ax) + (Cy-Ay)*(By-Ay);
	u /= (Bx-Ax)*(Bx-Ax) + (By-Ay)*(By-Ay);
	if (u<0) {
	 return (Ax-Cx)*(Ax-Cx) + (Ay-Cy)*(Ay-Cy);
	}
	if (u>1) {
	 return (Bx-Cx)*(Bx-Cx) + (By-Cy)*(By-Cy);
	}
	float Px = Ax + u*(Bx-Ax);
	float Py = Ay + u*(By-Ay);
	return (Px-Cx)*(Px-Cx) + (Py-Cy)*(Py-Cy);
}

__device__ inline float d_p_dist(float Cx, float Cy, float Ax, float Ay, float Bx, float By)
{
	float x = Bx-Ax, y = By-Ay;
        float u = (Cx-Ax)*x + (Cy-Ay)*y;
        u /= x*x + y*y;
	u = __saturatef (u); // 0.0 <= u <= 1.0
	float Px = Ax + u*x; // if u=0.0, Px= Ax, if u=1.0, Px = Bx, otherwise Px = Ax + u*(Bx-Ax)
	float Py = Ay + u*y;
	
	return (Px-Cx)*(Px-Cx) + (Py-Cy)*(Py-Cy);
}

__global__ void k_pdist(float *ddp, float *da, float *db, float x_nearest, float y_nearest, float x_new, float y_new, uint M) 
{
	for(uint i = threadIdx.x+blockDim.x*blockIdx.x; i<M; i+=gridDim.x*blockDim.x)
		ddp[i] = d_p_dist(da[i],db[i],x_nearest,y_nearest,x_new,y_new);
}

__global__ void k_pdist(float *ddp, float *da, float *db, float *dx_nearest, float *dy_nearest, float *dx_new, float *dy_new, uint M) 
{
	for(uint i = threadIdx.x+blockDim.x*blockIdx.x; i<M; i+=gridDim.x*blockDim.x)
		ddp[i] = d_p_dist(da[i],db[i],*dx_nearest,*dy_nearest,*dx_new,*dy_new);
}

void obstacle_free(RRT_PARAMS* params, RRT_VARS* vars)
{
	// compute distances from all obstacles to segment [(x_nearest,y_nearest),(x_new,y_new)]
	if (vars->index > THRES_GPU) 
		k_pdist<<<MIN(params->M/256+1,devp.num_multiproc*8),256>>>(devp.ddp, devp.da, devp.db, devp.dx_nearest, devp.dy_nearest, devp.dx_new, devp.dy_new, params->M);
	else
		k_pdist<<<MIN(params->M/256+1,devp.num_multiproc*8),256>>>(devp.ddp, devp.da, devp.db, vars->x_nearest, vars->y_nearest, vars->x_new, vars->y_new, params->M);
	CubDebugExit(hipDeviceSynchronize());

	// Compute minimun distance	
	CubDebugExit(hipcub::DeviceReduce::Min(devp.dcubtemp, devp.temp_bytes, devp.ddp, devp.dm, params->M));
	CubDebugExit(hipDeviceSynchronize());
	float m;
	CubDebugExit(hipMemcpy(&m,devp.dm,sizeof(float),hipMemcpyDeviceToHost));
	// collision if minimun distance is less than epsilon
	// variable collision has a value greater than 0 if collision
	vars->collision = params->epsilon - m;
}

void retrieve_solution(RRT_VARS* vars)
{
	if (vars->index > THRES_GPU) {
/*		CubDebugExit(hipMemcpy(vars->x+THRES_GPU+1,devp.dx+THRES_GPU+1,(vars->index-THRES_GPU)*sizeof(float),hipMemcpyDeviceToHost));
		CubDebugExit(hipMemcpy(vars->y+THRES_GPU+1,devp.dy+THRES_GPU+1,(vars->index-THRES_GPU)*sizeof(float),hipMemcpyDeviceToHost));
		CubDebugExit(hipMemcpy(vars->px+THRES_GPU+1,devp.dpx+THRES_GPU+1,(vars->index-THRES_GPU)*sizeof(float),hipMemcpyDeviceToHost));
		CubDebugExit(hipMemcpy(vars->py+THRES_GPU+1,devp.dpy+THRES_GPU+1,(vars->index-THRES_GPU)*sizeof(float),hipMemcpyDeviceToHost));
*/
		CubDebugExit(hipMemcpy(vars->x,devp.dx,(vars->index)*sizeof(float),hipMemcpyDeviceToHost));
		CubDebugExit(hipMemcpy(vars->y,devp.dy,(vars->index)*sizeof(float),hipMemcpyDeviceToHost));
		CubDebugExit(hipMemcpy(vars->px,devp.dpx,(vars->index)*sizeof(float),hipMemcpyDeviceToHost));
		CubDebugExit(hipMemcpy(vars->py,devp.dpy,(vars->index)*sizeof(float),hipMemcpyDeviceToHost));
	}
}

__global__ void k_extend_rrt (float *dx, float *dy, float *dpx, float *dpy, float *dx_new, float *dy_new, float *dx_nearest, float *dy_nearest, int index)
{
	dx[index] = *dx_new;
	dy[index] = *dy_new;
	dpx[index] = *dx_nearest;
	dpy[index] = *dy_nearest;
}

void extend_rrt(RRT_PARAMS* params, RRT_VARS* vars)
{
	if (vars->index <= THRES_GPU) {
		vars->x[vars->index] = vars->x_new;
		vars->y[vars->index] = vars->y_new;
		vars->px[vars->index] = vars->x_nearest;
		vars->py[vars->index] = vars->y_nearest;
	}

	if (vars->index == THRES_GPU) {
		CubDebugExit(hipMemcpy(devp.dx,vars->x,(vars->index+1)*sizeof(float),hipMemcpyHostToDevice));
		CubDebugExit(hipMemcpy(devp.dy,vars->y,(vars->index+1)*sizeof(float),hipMemcpyHostToDevice));
		CubDebugExit(hipMemcpy(devp.dpx,vars->px,(vars->index+1)*sizeof(float),hipMemcpyHostToDevice));
		CubDebugExit(hipMemcpy(devp.dpy,vars->py,(vars->index+1)*sizeof(float),hipMemcpyHostToDevice));
	}
	else if (vars->index > THRES_GPU) {
		k_extend_rrt<<<1,1>>>(devp.dx,devp.dy,devp.dpx,devp.dpy,devp.dx_new,devp.dy_new,devp.dx_nearest,devp.dy_nearest,vars->index);
		CubDebugExit(hipDeviceSynchronize());
		//hipMemcpy(devp.dx+vars->index,devp.dx_new,sizeof(float),hipMemcpyDeviceToDevice);
		//hipMemcpy(devp.dy+vars->index,devp.dy_new,sizeof(float),hipMemcpyDeviceToDevice);
		//hipMemcpy(devp.dx+vars->index,vars->x+vars->index,sizeof(float),hipMemcpyHostToDevice);
		//hipMemcpy(devp.dy+vars->index,vars->y+vars->index,sizeof(float),hipMemcpyHostToDevice);
	}
}


__global__ void k_rrts_pdistall(float *ddp, float *da, float *db, float *dx, float *dy, float *dx_new, float *dy_new, uint index, uint M) 
{
	float ldx_new=*dx_new, ldy_new=*dy_new; // preload data to registers
	for(uint i = threadIdx.x+blockDim.x*blockIdx.x; i<index; i+=gridDim.x*blockDim.x) {
		float ldx=dx[i],ldy=dy[i];  // preload data to registers
		for(uint j = threadIdx.y+blockDim.y*blockIdx.y; j<M; j+=gridDim.y*blockDim.y)
			ddp[i*M+j] = d_p_dist(da[j],db[j],ldx,ldy,ldx_new,ldy_new);
	}
}

/*
__global__ void k_rrts_lessepsilon(float *ddpp, float epsilon, int index) {
	for(uint i = threadIdx.x+blockDim.x*blockIdx.x; i<index; i+=gridDim.x*blockDim.x) 
		ddpp[i] =  (ddpp[i] < epsilon)? 1.f : 0.f;
}

__global__ void k_rrts_newcost(float* dcp, float *ddpp, float *ddc, float *dx, float *dy, float *dx_new, float *dy_new, int index) {
	for(uint i = threadIdx.x+blockDim.x*blockIdx.x; i<index; i+=gridDim.x*blockDim.x) {
		dcp[i] =  ddpp[i]>0.f ? INF : 
						ddc[i] + 
						(*dx_new - dx[i])*(*dx_new - dx[i]) +
						(*dy_new - dy[i])*(*dy_new - dy[i]);
	}
}
*/

// this kernel is the mix of the previous two
__global__ void k_rrts_lessepsilon_newcost(float* dcp, float *ddpp, float *dc, float *dx, float *dy, float *dx_new, float *dy_new, int index, float epsilon) {
	for(uint i = threadIdx.x+blockDim.x*blockIdx.x; i<index; i+=gridDim.x*blockDim.x) {
		dcp[i] =  ddpp[i]<epsilon ? INF : 
						dc[i] + 
						(*dx_new - dx[i])*(*dx_new - dx[i]) +
						(*dy_new - dy[i])*(*dy_new - dy[i]);
	}
}

__global__ void k_rrts_extend (float *dc, float *dx, float *dy, float *dpx, float *dpy, float *dx_new, float *dy_new, KeyValuePair<int,float>* d_argmin, int index)
{
	dx[index] = *dx_new;
	dy[index] = *dy_new;
	dpx[index] = dx[d_argmin->key];
	dpy[index] = dy[d_argmin->key];
	dc[index] = d_argmin->value;
}

__global__ void k_rrts_fixedges (float *dc, float *dx, float *dy, float *dpx, float *dpy, float *dx_new, float *dy_new, KeyValuePair<int,float>* d_argmin, int index) 
{
	for(uint i = threadIdx.x+blockDim.x*blockIdx.x; i<index; i+=gridDim.x*blockDim.x) {
		float aux = dc[index] + 
					(*dx_new - dx[i])*(*dx_new - dx[i]) +
					(*dy_new - dy[i])*(*dy_new - dy[i]);
		
		if (dc[i] > aux) {
			dpx[i] = *dx_new;
			dpy[i] = *dy_new;
			dc[i] = aux;
		}
	}
}

void extend_rrt_star(RRT_PARAMS* params, RRT_VARS* vars)
{
	if (vars->index <= THRES_GPU) {
		// compute squared distances from all obstacles to all segments [(x,y),(x_new,y_new)] where (x,y) are points in RRT
		#pragma omp parallel for collapse(2)
		for (int i=0;i<vars->index;i++) {
			for (int j=0;j<params->M;j++) {
				vars->dpp[i* params->M + j] = p_dist(params->a[j],params->b[j],vars->x[i],vars->y[i],vars->x_new,vars->y_new);
			}
		}
	}
	else {
		dim3 block_size(16,16);
		dim3 grid_size(MIN(vars->index/block_size.x+1,devp.num_multiproc*4),MIN(params->M/block_size.y+1,devp.num_multiproc*4));
		k_rrts_pdistall<<<grid_size,block_size>>>(devp.ddpp, devp.da, devp.db, devp.dx, devp.dy, devp.dx_new, devp.dy_new, vars->index, params->M);
		CubDebugExit(hipDeviceSynchronize());
	}
	
	if (vars->index <= THRES_GPU) {
		// For each point (x,y) in RRT, compute the minimun distance 
		#pragma omp parallel for
		for (int i=0;i<vars->index;i++) {
			float m = INF;
			#pragma omp parallel for reduction(min:m)
			for (int j=0;j<params->M;j++) {
				if (vars->dpp[i*params->M+j] < m) {
					m = vars->dpp[i*params->M+j];
				}
			}
			// if the minimun distance is less than epsilon, set a flag to avoid this possible edge
			if (m< params->epsilon) {
				vars->dpp[i*params->M]=1;
			} else {
				vars->dpp[i*params->M]=0;
			}
		}
	}
	else {
		// For each point (x,y) in RRT, compute the minimun distance 
		// Using CUB segmented reduce, a segment per row in the matrix vars->index*params->M
		CubDebugExit(hipcub::DeviceSegmentedReduce::Min(devp.dcubtemp, devp.temp_bytes, devp.ddpp, devp.dd,
			vars->index, devp.dsegments, devp.dsegments + 1));
		CubDebugExit(hipDeviceSynchronize());

		// if the minimun distance is less than epsilon, set a flag to avoid this possible edge
		// this is done in the following step
		//k_rrts_lessepsilon<<<MIN(vars->index/256+1,devp.num_multiproc*8),256>>>(devp.dd,params->epsilon,vars->index);
		//CubDebugExit(hipDeviceSynchronize());
	}
	
	if (vars->index <= THRES_GPU) {
		// compute new cost for all points in RRT 
		#pragma omp parallel for
		for (int i=0;i<vars->index;i++) {
			vars->cp[i] =  vars->dpp[i*params->M]>0 ? INF : 
								vars->c[i] + 
									(vars->x_new - vars->x[i])*(vars->x_new - vars->x[i]) +
									(vars->y_new - vars->y[i])*(vars->y_new - vars->y[i]);
		}
	}
	else {
		// compute new cost for all points in RRT
		k_rrts_lessepsilon_newcost<<<MIN(vars->index/256+1,devp.num_multiproc*8),256>>>(devp.dcp, devp.dd, devp.dc, devp.dx, devp.dy, devp.dx_new, devp.dy_new, vars->index, params->epsilon);
		CubDebugExit(hipDeviceSynchronize());
	}

	if (vars->index <= THRES_GPU) {
		// compute minimun cost	
		XYD value = {0,0,INF};
		#pragma omp parallel for reduction(xyd_min:value)
		for (int i=0;i<vars->index;i++) {
			XYD new_value = {vars->x[i],vars->y[i],vars->cp[i]};
			value = xyd_min2(value,new_value);
		}

		// extend RRT*
		vars->x[vars->index] = vars->x_new;
		vars->y[vars->index] = vars->y_new;
		vars->px[vars->index] = value.x;
		vars->py[vars->index] = value.y;
		vars->c[vars->index] = value.d;

		// Fix edges
		#pragma omp parallel for
		for (int i=0;i<vars->index;i++) {
			float aux = vars->c[vars->index] + 
							(vars->x_new - vars->x[i])*(vars->x_new - vars->x[i]) +
							(vars->y_new - vars->y[i])*(vars->y_new - vars->y[i]);
							
			if (vars->c[i] > aux) {
				vars->px[i] = vars->x_new;
				vars->py[i] = vars->y_new;
				vars->c[i] = aux;
			}
		}
	}
	else {
		// compute minimun cost
		CubDebugExit(hipcub::DeviceReduce::ArgMin(devp.dcubtemp, devp.temp_bytes, devp.dcp, devp.d_argmin, vars->index));
		CubDebugExit(hipDeviceSynchronize());

		// extend RRT*
		k_rrts_extend<<<1,1>>> (devp.dc, devp.dx, devp.dy, devp.dpx, devp.dpy, devp.dx_new, devp.dy_new, devp.d_argmin, vars->index);
		CubDebugExit(hipDeviceSynchronize());

		// Fix edges
		k_rrts_fixedges<<<MIN(vars->index/256+1,devp.num_multiproc*8),256>>> (devp.dc, devp.dx, devp.dy, devp.dpx, devp.dpy, devp.dx_new, devp.dy_new, devp.d_argmin, vars->index);
		CubDebugExit(hipDeviceSynchronize());
	}

	// reintegrate partial results from CPU to GPU
	if (vars->index == THRES_GPU) {
		CubDebugExit(hipMemcpy(devp.dx,vars->x,(vars->index+1)*sizeof(float),hipMemcpyHostToDevice));
		CubDebugExit(hipMemcpy(devp.dy,vars->y,(vars->index+1)*sizeof(float),hipMemcpyHostToDevice));
		CubDebugExit(hipMemcpy(devp.dpx,vars->px,(vars->index+1)*sizeof(float),hipMemcpyHostToDevice));
		CubDebugExit(hipMemcpy(devp.dpy,vars->py,(vars->index+1)*sizeof(float),hipMemcpyHostToDevice));
		CubDebugExit(hipMemcpy(devp.dc,vars->c,(vars->index+1)*sizeof(float),hipMemcpyHostToDevice));
	}
}

void enps_rrt_one_iteration(RRT_PARAMS* params, RRT_VARS* vars)
{
	// Exit if halting condition has been reached
	if (vars->halt) {		
		return;
	}
	
	// set collision = 0
	vars->collision = 0;
	
	// compute (x_rand, y_rand)
	vars->x_rand = params->p * rnd();
	vars->y_rand = params->q * rnd();
	
	// compute (x_nearest, y_nearest)
	nearest(params, vars);
	
	// compute (x_new, y_new)
	if (vars->index <= THRES_GPU) {
		vars->x_new = vars->x_nearest + params->delta * (vars->x_rand - vars->x_nearest) / sqrt(vars->d[0]);
		vars->y_new = vars->y_nearest + params->delta * (vars->y_rand - vars->y_nearest) / sqrt(vars->d[0]);
	}

	// compute obstacle collision
	obstacle_free(params, vars);
	
	// Exit if collision from (x_nearest, y_nearest) to (x_new, y_new)
	if (vars->collision > 0) {
		return;
	}	
	
	// Extend RRT tree
	switch(params->algorithm)
	{
		case RRT_ALGORITHM:
			extend_rrt(params,vars);
		break;
		case RRT_STAR_ALGORITHM:
			extend_rrt_star(params,vars);
		break;
		default:
		;
	}
	
	// Increment RRT node index
	vars->index++;
	
	// If node index is 2^n then halt
	if (vars->index == params->N) {
		vars->halt = 1;
		retrieve_solution(vars);
	}
}


