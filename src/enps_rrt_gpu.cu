#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

extern "C" { // nvcc compiles en C++
#include <enps_rrt.h>
#include <pgm.h>
}

#include <omp.h>
#include <hip/hip_runtime.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

using namespace std;

void init_params(const char* file, int n, float delta, int debug, int algorithm, RRT_PARAMS* params)
{
	PGM* map = load_pgm(file);
	params->map = load_pgm(file);

	remove_inner_obstacles(map);
	
	params->epsilon = ROBOT_RADIUS * ROBOT_RADIUS;
	params->delta =   delta;
	
	params->n = n;
	params->N = 1<<n; // Number of nodes in RRT
	
	params->debug = debug;
	
	params->p = map->width * RESOLUTION;
	params->q = map->height * RESOLUTION;
	
	params->algorithm = algorithm;
	
	float x=0,y=0;
	int c=0;
	params->a = (float*)malloc(map->width * map->height * sizeof(float));
	params->b = (float*)malloc(map->width * map->height * sizeof(float));
	
	for (int i=0;i<map->height;i++) {
		for (int j=0;j<map->width;j++) {
			x+=RESOLUTION;
			if (IS_OBSTACLE(map,i,j)){
				params->a[c] = x;
				params->b[c] = y;
				c++;
			}
		}
		y += RESOLUTION;
		x = 0;
	}
	params->m = 0;
	params->M = 1;
	
	while (params->M < c) {
		params->m++;
		params->M <<= 1;
	}
	if (debug) {
		printf("Map: %s\n",file);
		printf("Number of obstacles: %d\n",c);
		printf("Number of nodes: %d\n",params->M);
	}
	params->a = (float*)realloc(params->a, (params->M)*sizeof(float));
	params->b = (float*)realloc(params->b, (params->M)*sizeof(float));
	//free(params->a);
	//free(params->b);
	//hipMallocManaged(&params->a,(params->M)*sizeof(float));
	//hipMallocManaged(&params->b,(params->M)*sizeof(float));
	for (int i=c;i<params->M;i++) {
		params->a[i] = 3* params->p;
		params->b[i] = 3* params->q;
	}
	params->device = -1;
  	hipGetDevice(&params->device);
/*  	hipMemPrefetchAsync(params->a, params->M*sizeof(float), params->device, NULL);
  	hipMemPrefetchAsync(params->b, params->M*sizeof(float), params->device, NULL);
*/
	destroy_pgm(map);
}


void init_vars(float x_init, float y_init, const RRT_PARAMS* params, RRT_VARS* vars)
{
	//vars->x = (float*)malloc((params->N)*sizeof(float));
	//vars->y = (float*)malloc((params->N)*sizeof(float));
	hipMallocManaged(&vars->x,(params->N)*sizeof(float));
	hipMallocManaged(&vars->y,(params->N)*sizeof(float));
	
	vars->x[0] = x_init;
	vars->y[0] = y_init;
	
	for (int i=1; i< params->N; i++) {
		vars->x[i] = 3* params->p;
		vars->y[i] = 3* params->q;
	}
	
	vars->px = (float*)malloc((params->N)*sizeof(float));
	vars->py = (float*)malloc((params->N)*sizeof(float));
	//hipMallocManaged(&vars->x,(params->N)*sizeof(float));
    //hipMallocManaged(&vars->y,(params->N)*sizeof(float));
	
	//vars->d = (float*)malloc((params->N)*sizeof(float));
	hipMallocManaged(&vars->d,(params->N)*sizeof(float));
			
	vars->dp = (float*)malloc((params->M)*sizeof(float));
	//hipMallocManaged(&vars->dp,(params->M)*sizeof(float));

	// consider using hipMemPrefetchAsync() for better performance
	vars->x_rand = 0;
	vars->y_rand = 0;
	
	vars->x_new = 0;
	vars->y_new = 0;
	
	vars->x_nearest = 0;
	vars->y_nearest = 0;
	
	vars->collision = 0;
	
	vars->index = 1;
	vars->halt = 0;
	
	if (params->algorithm == RRT_STAR_ALGORITHM) {
		vars->dpp = (float*)malloc(params->M * params->N * sizeof(float));
		vars->c = (float*)malloc(params->N * sizeof(float));
		vars->cp = (float*)malloc(params->N * sizeof(float));
		vars->c[0] = 0;
	} 
}

void free_memory(RRT_PARAMS* params,RRT_VARS* vars)
{
	destroy_pgm(params->map);
	free(params->a);
	free(params->b);
	//free(vars->x);
	//free(vars->y);
	free(vars->px);
	free(vars->py);
	//free(vars->d);
	free(vars->dp);

	//hipFree(params->a);
	//hipFree(params->b);
	hipFree(vars->x);
	hipFree(vars->y);
	//hipFree(vars->px);
	//hipFree(vars->py);
	hipFree(vars->d);
	//hipFree(vars->dp);
	
	if (params->algorithm==RRT_STAR_ALGORITHM) {
		free(vars->dpp);
		free(vars->c);
		free(vars->cp);
	}
}


float rnd()
{
	return (float)rand()/(float)(RAND_MAX);
}


// Squared distance from point (Cx,Cy) to segment [(Ax,Ay),(Bx,By)]
__host__ __device__ float p_dist(float Cx, float Cy, float Ax, float Ay, float Bx, float By)
{
	float u = (Cx-Ax)*(Bx-Ax) + (Cy-Ay)*(By-Ay);
	u /= (Bx-Ax)*(Bx-Ax) + (By-Ay)*(By-Ay);
	if (u<0) {
	 return (Ax-Cx)*(Ax-Cx) + (Ay-Cy)*(Ay-Cy);
	}
	if (u>1) {
	 return (Bx-Cx)*(Bx-Cx) + (By-Cy)*(By-Cy);
	}
	float Px = Ax + u*(Bx-Ax);
	float Py = Ay + u*(By-Ay);
	return (Px-Cx)*(Px-Cx) + (Py-Cy)*(Py-Cy);
}

struct d_p_dist : public thrust::binary_function<float,float,float>
{

	float ax,ay,bx,by;
	__host__ __device__ float operator()(float cx, float cy) { return p_dist(cx,cy,ax,ay,bx,by); }
};

void obstacle_free(RRT_PARAMS* params, RRT_VARS* vars)
{
	//k_obstacle_free<<<256,params->M/256+1>>>(vars->dp,params->a,params->b,vars->x_nearest,vars->y_nearest,vars->x_new,vars->y_new);
/*
	hipMemPrefetchAsync(params->a, params->M*sizeof(float), params->device, NULL);
  	hipMemPrefetchAsync(params->b, params->M*sizeof(float), params->device, NULL);
  	hipDeviceSynchronize();

	// compute distances from all obstacles to segment [(x_nearest,y_nearest),(x_new,y_new)]
	struct d_p_dist dpdist_op;
	dpdist_op.ax=vars->x_nearest;
	dpdist_op.ay=vars->y_nearest;
	dpdist_op.bx=vars->x_new;
	dpdist_op.by=vars->y_new;
	thrust::transform(thrust::device, params->a, params->a + params->M, params->b, vars->dp, dpdist_op); 

	float * deb = new float[params->M];	
	for (int i=0;i<	params->M;i++) {
		deb[i] = p_dist(params->a[i],params->b[i],vars->x_nearest,vars->y_nearest,vars->x_new,vars->y_new);
	}


	/*hipDeviceSynchronize();
	hipMemPrefetchAsync(vars->dp,params->M*sizeof(float),0);
	hipDeviceSynchronize();
	for (int i =0; i< params->M; i++) {
		if (deb[i] != vars->dp[i]) {
			printf("GPU[%d]=%f != CPU[%d]=%f \n",i,vars->dp[i],i,deb[i]);
		}
	}
	float m,m1 = INF;	
	for (int i=0;i<params->M;i++) {
		if (deb[i]<m1) {
			m1 = deb[i];
		}
	}
	/*if (m!=m1){
		printf("mGPU=%f != mCPU=%f \n",m,m1);		
	}
	delete [] deb;

	// Compute minimun distance
	//float *m_pos = thrust::min_elem(thrust::device, vars->dp, vars->dp + params->M);
	//float m = *m_pos;
	//float m = thrust::reduce(thrust::device, vars->dp, vars->dp + params->M, INF, thrust::minimum<float>());
	m=m1;

	// collision if minimun distance is less than epsilon
	// variable collision has a value greater than 0 if collision
	vars->collision = params->epsilon - m;
*/

	// CPU version:
	//compute distances from all obstacles to segment [(x_nearest,y_nearest),(x_new,y_new)]
	#pragma omp parallel for
	for (int i=0;i<	params->M;i++) {
		vars->dp[i] = p_dist(params->a[i],params->b[i],vars->x_nearest,vars->y_nearest,vars->x_new,vars->y_new);
	}
	
	// Compute minimun distance
	float m = INF;
	#pragma omp parallel for reduction(min:m)
	for (int i=0;i<params->M;i++) {
		if (vars->dp[i]<m) {
			m = vars->dp[i];
		}
	}
	// collision if minimun distance is less than epsilon
	// variable collision has a value greater than 0 if collision
	vars->collision = params->epsilon - m;
}


XYD xyd_min2(XYD a, XYD b)
{
	return a.d < b.d ? a : b;
}


#pragma omp declare reduction(xyd_min : XYD : omp_out=xyd_min2(omp_out,omp_in))\
		initializer(omp_priv={0,0,INF})


struct d_squared_dist : public thrust::binary_function<float,float,float>
{
	float xr,yr;
	__host__ __device__ float operator()(float x, float y) { return (x - xr) * (x - xr) + (y - yr) * (y - yr); }
};

void nearest(RRT_PARAMS* params, RRT_VARS* vars)
{
	// compute squared distances from all points in RRT to (x_rand,y_rand)
	struct d_squared_dist dsqdist_op;
    dsqdist_op.xr=vars->x_rand;
	dsqdist_op.yr=vars->y_rand;
	thrust::transform(thrust::device, vars->x, vars->x + vars->index, vars->y, vars->d, dsqdist_op);

	float *m_pos = thrust::min_element(thrust::device, vars->d, vars->d + vars->index);
	int pos = m_pos - vars->d;
	vars->x_nearest = vars->x[pos];
	vars->y_nearest = vars->y[pos];
	vars->d[0] = *m_pos;	

	// compute squared distances from all points in RRT to (x_rand,y_rand)
	/*#pragma omp parallel for
	for (int i=0;i<vars->index;i++) {
		vars->d[i] = (vars->x[i] - vars->x_rand) * (vars->x[i] - vars->x_rand) +
						(vars->y[i] - vars->y_rand) * (vars->y[i] - vars->y_rand);
	}	

	// compute minimun distance and nearest point
	XYD value = {0,0,INF};
	#pragma omp parallel for reduction(xyd_min:value)
	for (int i=0;i<vars->index;i++) {
		XYD new_value = {vars->x[i],vars->y[i],vars->d[i]};
		value = xyd_min2(value,new_value);
	}
	vars->x_nearest = value.x;
	vars->y_nearest = value.y;
	vars->d[0] = value.d;*/
}



void extend_rrt(RRT_PARAMS* params, RRT_VARS* vars)
{
	vars->x[vars->index] = vars->x_new;
	vars->y[vars->index] = vars->y_new;
	vars->px[vars->index] = vars->x_nearest;
	vars->py[vars->index] = vars->y_nearest;
}


void extend_rrt_star(RRT_PARAMS* params, RRT_VARS* vars)
{
	// compute squared distances from all obstacles to all segments [(x,y),(x_new,y_new)] where (x,y) are points in RRT
	#pragma omp parallel for collapse(2)
	for (int i=0;i<vars->index;i++) {
		for (int j=0;j<params->M;j++) {
			vars->dpp[i* params->M + j] = p_dist(params->a[j],params->b[j],vars->x[i],vars->y[i],vars->x_new,vars->y_new);
		}
	}
	
	// For each point (x,y) in RRT, compute the minimun distance 
	#pragma omp parallel for
	for (int i=0;i<vars->index;i++) {
		float m = INF;
		#pragma omp parallel for reduction(min:m)
		for (int j=0;j<params->M;j++) {
			if (vars->dpp[i*params->M+j] < m) {
				m = vars->dpp[i*params->M+j];
			}
		}
		// if the minimun distance is less than epsilon, set a flag to avoid this possible edge
		if (m< params->epsilon) {
			vars->dpp[i*params->M]=1;
		} else {
			vars->dpp[i*params->M]=0;
		}
	}
	
	// compute new cost for all points in RRT 
	#pragma omp parallel for
	for (int i=0;i<vars->index;i++) {
		vars->cp[i] =  vars->dpp[i*params->M]>0 ? INF : 
							vars->c[i] + 
								(vars->x_new - vars->x[i])*(vars->x_new - vars->x[i]) +
								(vars->y_new - vars->y[i])*(vars->y_new - vars->y[i]);
		
		
		//vars->cp[i] = vars->c[i] + 
		//				(vars->x_new - vars->x[i])*(vars->x_new - vars->x[i]) +
		//				(vars->y_new - vars->y[i])*(vars->y_new - vars->y[i]) +
		//				INF * vars->dpp[i*params->M]; 
	}
	
	// compute minimun cost	
	XYD value = {0,0,INF};
	#pragma omp parallel for reduction(xyd_min:value)
	for (int i=0;i<vars->index;i++) {
		XYD new_value = {vars->x[i],vars->y[i],vars->cp[i]};
		value = xyd_min2(value,new_value);
	}
	
	vars->x[vars->index] = vars->x_new;
	vars->y[vars->index] = vars->y_new;
	vars->px[vars->index] = value.x;
	vars->py[vars->index] = value.y;
	vars->c[vars->index] = value.d;

	// Fix edges
	#pragma omp parallel for
	for (int i=0;i<vars->index;i++) {
		float aux = vars->c[vars->index] + 
						(vars->x_new - vars->x[i])*(vars->x_new - vars->x[i]) +
						(vars->y_new - vars->y[i])*(vars->y_new - vars->y[i]);
						
		if (vars->c[i] > aux) {
			vars->px[i] = vars->x_new;
			vars->py[i] = vars->y_new;
			vars->c[i] = aux;
		}
	}
}

void enps_rrt_one_iteration(RRT_PARAMS* params, RRT_VARS* vars)
{
	// Exit if halting condition has been reached
	if (vars->halt) {
		return;
	}
	
	// set collision = 0
	vars->collision = 0;
	
	// compute (x_rand, y_rand)
	vars->x_rand = params->p * rnd();
	vars->y_rand = params->q * rnd();
	
	// compute (x_nearest, y_nearest)
	nearest(params, vars);
	
	// compute (x_new, y_new)
	vars->x_new = vars->x_nearest + params->delta * (vars->x_rand - vars->x_nearest) / sqrt(vars->d[0]);
	vars->y_new = vars->y_nearest + params->delta * (vars->y_rand - vars->y_nearest) / sqrt(vars->d[0]);
	
	// compute obstacle collision
	obstacle_free(params, vars);
	
	// Exit if collision from (x_nearest, y_nearest) to (x_new, y_new)
	if (vars->collision > 0) {
		return;
	}
	cout << "here" << endl;
	
	// Extend RRT tree
	switch(params->algorithm)
	{
		case RRT_ALGORITHM:
			extend_rrt(params,vars);
		break;
		case RRT_STAR_ALGORITHM:
			extend_rrt_star(params,vars);
		break;
		default:
		;
	}
	
	// Increment RRT node index
	vars->index++;
	
	// If node index is 2^n then halt
	if (vars->index == params->N) {
		vars->halt = 1;
	}
}

